
#include <hip/hip_runtime.h>
#include <string>
#include <vector>
#include <iostream>
#include <sstream>
#include <fstream>
#include <iterator>


#define gpuErrchk(ans) { gpuAssert( (ans), __FILE__, __LINE__ ); }

inline void
gpuAssert( hipError_t code, const char * file, int line, bool abort = true )
{
	if ( hipSuccess != code )
	{
		fprintf( stderr, "\nGPUassert: %s %s %d\n", hipGetErrorString( code ), file, line );
		if ( abort )
			exit( code );
	}


	return;

}

namespace cuda_kernels{
	__host__ __device__ int index_from_coordinates(int column, int row, int level, int length, int width, int heighth){
		return (column % length) + (row % width) * length + (level % heighth) * length * width;
	}
	__host__ __device__ int calc_neighbours(int* grid, int column, int row, int level, int length, int width, int heighth){
		return grid[index_from_coordinates(column + -1, row + -1, level + -1, length, width, heighth)] + grid[index_from_coordinates(column + -1, row + -1, level + 0, length, width, heighth)] + grid[index_from_coordinates(column + -1, row + -1, level + 1, length, width, heighth)] + grid[index_from_coordinates(column + -1, row + 0, level + -1, length, width, heighth)] + grid[index_from_coordinates(column + -1, row + 0, level + 0, length, width, heighth)] + grid[index_from_coordinates(column + -1, row + 0, level + 1, length, width, heighth)] + grid[index_from_coordinates(column + -1, row + 1, level + -1, length, width, heighth)] + grid[index_from_coordinates(column + -1, row + 1, level + 0, length, width, heighth)] + grid[index_from_coordinates(column + -1, row + 1, level + 1, length, width, heighth)] + grid[index_from_coordinates(column + 0, row + -1, level + -1, length, width, heighth)] + grid[index_from_coordinates(column + 0, row + -1, level + 0, length, width, heighth)] + grid[index_from_coordinates(column + 0, row + -1, level + 1, length, width, heighth)] + grid[index_from_coordinates(column + 0, row + 0, level + -1, length, width, heighth)] + grid[index_from_coordinates(column + 0, row + 0, level + 1, length, width, heighth)] + grid[index_from_coordinates(column + 0, row + 1, level + -1, length, width, heighth)] + grid[index_from_coordinates(column + 0, row + 1, level + 0, length, width, heighth)] + grid[index_from_coordinates(column + 0, row + 1, level + 1, length, width, heighth)] + grid[index_from_coordinates(column + 1, row + -1, level + -1, length, width, heighth)] + grid[index_from_coordinates(column + 1, row + -1, level + 0, length, width, heighth)] + grid[index_from_coordinates(column + 1, row + -1, level + 1, length, width, heighth)] + grid[index_from_coordinates(column + 1, row + 0, level + -1, length, width, heighth)] + grid[index_from_coordinates(column + 1, row + 0, level + 0, length, width, heighth)] + grid[index_from_coordinates(column + 1, row + 0, level + 1, length, width, heighth)] + grid[index_from_coordinates(column + 1, row + 1, level + -1, length, width, heighth)] + grid[index_from_coordinates(column + 1, row + 1, level + 0, length, width, heighth)] + grid[index_from_coordinates(column + 1, row + 1, level + 1, length, width, heighth)];
	}

	__global__ void calc_next_generation_all_global(int* current_grid, int* next_grid, int length, int width, int heighth){ 
		// Call to global memory approx 28 times per cell - very slow
		int column = blockIdx.x * blockDim.x + threadIdx.x;
		int row = blockIdx.y * blockDim.y + threadIdx.y;
		int level = blockIdx.z * blockDim.z + threadIdx.z;

		int place = index_from_coordinates(column, row, level, length, width, heighth);

		if (column < length and row < width and level < heighth)
		{
			int state = current_grid[place]; // slow
			int neighbours = calc_neighbours(current_grid, column, row, level, length, width, heighth); // 26 * slow

			next_grid[place] = 0; // slow
			if (state == 0 and (neighbours == 6 or neighbours == 7))
			{
				next_grid[place] = 1; // slow
			}
			if (state == 1 and (neighbours == 4 or neighbours == 5 or neighbours == 6 or neighbours == 7))
			{
				next_grid[place] = 1; // slow
			}
		}
	}

	__global__ void calc_next_generation_shared_areas(int* current_grid, int* next_grid, int length, int width, int heighth){
		// Call to global memory approx 2 times per cell - fast
		//copy areas for each block into it's shared memory and then calculate neighbours
		int column = blockIdx.x * blockDim.x + threadIdx.x - 1;
		int row = blockIdx.y * blockDim.y + threadIdx.y - 1;
		int level = blockIdx.z * blockDim.z + threadIdx.z - 1;

		if (column < length + 2 and row < width + 2 and level < heighth + 2)
		{
			int place = index_from_coordinates(column, row, level, length, width, heighth);

			__shared__ int area[8192]; // 32kb of shared memory

			area[threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y] = current_grid[place]; // slow

			__syncthreads();

			// now we can calculate neighbours fast in a slightly smaller area
			if ((threadIdx.x > 0 and threadIdx.x < blockDim.x - 1 and threadIdx.y > 0 and threadIdx.y < blockDim.y - 1 and threadIdx.z > 0 and threadIdx.z < blockDim.z - 1) or column > length or row > width or level > heighth)
			{
				int neighbours;
				neighbours = calc_neighbours(area, threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y, blockDim.z); // fast

				int state = area[threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y];
				
				int newstate = 0;
				if (state == 0 and (neighbours == 6 or neighbours == 7))
				{
					newstate = 1;
				}
				if (state == 1 and (neighbours == 4 or neighbours == 5 or neighbours == 6 or neighbours == 7))
				{
					newstate = 1;
				}

				next_grid[place] = newstate; //slow
			}
		}
	}
}

void read_size(std::string filename, int* length, int* width, int* heighth){
	std::ifstream data(filename);
	std::string line;
	std::getline(data, line);

	std::stringstream lineStream(line);
	std::vector<std::string> parsedRow;
	std::string cell;

	while(std::getline(lineStream, cell, ' '))
    {
        parsedRow.push_back(cell);
    }

    *length = std::stoi(parsedRow[0]);
    *width = std::stoi(parsedRow[1]);
    *heighth = std::stoi(parsedRow[2]);
}

void read_input(std::string filename, int* state){
	std::ifstream data(filename);
	std::string line;
	std::getline(data, line);

	std::stringstream lineStream(line);
	std::vector<std::string> parsedRow;
	std::string cell;

	while(std::getline(lineStream, cell, ' '))
    {
        parsedRow.push_back(cell);
    }

    int length = std::stoi(parsedRow[0]);
    int width = std::stoi(parsedRow[1]);
    int heighth = std::stoi(parsedRow[2]);

    while(std::getline(data,line))
    {
        std::stringstream lineStream(line);
        std::string cell;
        std::vector<std::string> parsedRow;
        while(std::getline(lineStream, cell ,' '))
        {
            parsedRow.push_back(cell);
        }
        if (std::stoi(parsedRow[0]) != 0 or std::stoi(parsedRow[1]) != 0 or std::stoi(parsedRow[2]) != 0 or std::stoi(parsedRow[3]) != 0)
        {
        	state[std::stoi(parsedRow[1]) + std::stoi(parsedRow[2]) * length + std::stoi(parsedRow[3]) * length * width] = std::stoi(parsedRow[0]);
        }
    }
}

void append_state_to_file(std::string filename, int* field_d, int length, int width, int heighth){
	int field_h[length * width * heighth];
	size_t size = length * width * heighth * sizeof(int);
	hipMemcpy(field_h, field_d, size, hipMemcpyDeviceToHost);

	std::ofstream file;
	file.open(filename, std::ios_base::app);

	for (int i = 0; i < length; ++i)
	{
		for (int j = 0; j < width; ++j)
		{
			for (int k = 0; k < heighth; ++k)
			{
				int place = i + j * length + k * length * width;
				if (field_h[place] != 0){
					file << field_h[place] << " " << i << " " << j << " " << k << "\n"; 
				}
			}
		}
	}
	file << "0 0 0 0\n";
}

int main(){
	std::string filename;
	std::cin >> filename;
	int length = 20, width = 20, heighth = 20;

	read_size(filename, &length, &width, &heighth);

	int field_h[length * width * heighth];

	for (int i = 0; i < length; ++i)
	{
		for (int j = 0; j < width; ++j)
		{
			for (int k = 0; k < heighth; ++k)
			{
				field_h[i + j * length + k * length * width] = 0;
			}
		}
	}

	read_input(filename, field_h);

	int* field0_d;
	int* field1_d;
	size_t size = length * width * heighth * sizeof(int);
	gpuErrchk(hipMalloc(&field0_d, size));
	gpuErrchk(hipMalloc(&field1_d, size));

	gpuErrchk(hipMemcpy(field0_d, field_h, size, hipMemcpyHostToDevice));

	std::ofstream ofs;
	ofs.open("test.out", std::ofstream::out | std::ofstream::trunc);
	ofs.close();

	dim3 tpb(10, 10, 10);
	dim3 bpg(length / tpb.x + 1, width / tpb.y + 1, heighth / tpb.z + 1);



	for (int i = 0; i < 10; ++i)
	{
		append_state_to_file("test.out", field0_d, length, width, heighth);
		cuda_kernels::calc_next_generation_all_global<<<bpg, tpb>>>(field0_d, field1_d, length, width, heighth);
		gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

		std::swap(field0_d, field1_d);
	}

return 0;
}